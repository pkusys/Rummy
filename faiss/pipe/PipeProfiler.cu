#include "hip/hip_runtime.h"
/**
 * Copyright (c) zili zhang & fangyue liu @PKU.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <vector>
#include <random>
#include <sys/time.h>
#include <stdio.h>
#include <faiss/pipe/PipeProfiler.cuh>

// Record the current time (ms)
double timepoint() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec * 1e3 + tv.tv_usec * 1e-3;
}

namespace faiss{
namespace gpu{




PipeProfiler::PipeProfiler(IndexIVFPipe *index)
    {
        index_ = index;
        pc_ = index->pipe_cluster;
        pgr_ = index->pipe_provider;
        maxDataCnt = 4096 * 8 * 2;
        nqMax = 1024;
        trans = new TranProfiler(this);
        coms = new ComProfiler(this);
    }

void PipeProfiler::train(){
    // Train the sub-profilers
    coms->train();
    trans->train();

    istrained = true;
}

void PipeProfiler::save(const char* path_){
    char path[100];
    if(strcmp(path_, "") == 0 ){
        strcpy(path, "profileSave.txt");
    }
    else{
        strcpy(path, path_);
    }

    FILE * fp;

    fp = fopen (path, "w+");

    fprintf(fp, "{trans}\n");
    for (auto it = trans->tranTimeDict.begin(); it != trans->tranTimeDict.end(); it++){
        fprintf(fp, "%d %lf\n", it->first, it->second);
    }
    fprintf(fp, "%zu %lf\n", (unsigned long)0, 0.);

    fprintf(fp, "{coms}\n");
    for (auto it = coms->computeTimeDict.begin(); it != coms->computeTimeDict.end(); it++){
        fprintf(fp, "%d %lf\n", it->first, it->second);   
    }
    fprintf(fp,"%d %lf\n", 0, 0.);
    
    fprintf(fp,"{the-end}\n");

    fclose(fp);

}

void PipeProfiler::load(const char* path_){

    char path[100];
    char buffer[100];
    if(strcmp(path_, "") == 0 ){
        strcpy(path, "profileSave.txt");
    }
    else{
        strcpy(path, path_);
    }

    coms->computeTimeDict.clear();
    trans->tranTimeDict.clear();
    FILE * fp;

    fp = fopen (path, "r");
    fscanf(fp, "%s", buffer);
    // printf("loading profiler starts.\n");
    // printf("%s\n",buffer);

    while(true){
        int key;
        double value;
        fscanf(fp, "%d %lf", &key, &value);
        // printf("%d %lf\n", key, value);
        if(key == 0){
            break;
        }
        trans->tranTimeDict[key] = value;
    }        
    fscanf(fp, "%s", buffer);
    // printf("%s\n", buffer);

    while(true){
        int key;
        double value;
        fscanf(fp, "%d %lf", &key, &value);
        if(key == 0){
            break;
        }
        coms->computeTimeDict[key] = value;
    }
    
    fscanf(fp, "%s", buffer);
    // printf("%s\n", buffer);
    fclose(fp);

    trans->istrained = true;
    coms->istrained = true;
    this->istrained = true;

    return;
}

double PipeProfiler::queryTran(int pageCnt) {
    FAISS_ASSERT(trans->istrained);

    auto target = trans->tranTimeDict.find(pageCnt);

    if (target != trans->tranTimeDict.end()){
        return target->second;
    }

    auto up = trans->tranTimeDict.lower_bound(pageCnt);
    auto down = up;
    if(up == trans->tranTimeDict.end()){
        up--;
        down = up;
        down--;
    }
    else if (down == trans->tranTimeDict.begin()){
        up ++;
    }
    else{
        down --;
    }


    double downTime = down->second;
    double upTime = up->second;
    double realTime = downTime + (upTime - downTime) * (pageCnt - (double)(down->first)) / ((double)(up->first) - (double)(down->first));


    return realTime;
}

double PipeProfiler::queryCom(int dataCnt) {
    FAISS_ASSERT(coms->istrained);

    auto target = coms->computeTimeDict.find(dataCnt);
    if(target != coms->computeTimeDict.end()){
        return target->second;
    }
    auto up = coms->computeTimeDict.lower_bound(dataCnt);
    auto down = up;
    if(up == coms->computeTimeDict.begin()){
        up++;
        FAISS_ASSERT(up != coms->computeTimeDict.end());
    }
    else{
        if(up == coms->computeTimeDict.end()){
            up--;
            down = up;
        }
        down--;
    }
    double upTime = up->second;
    double downTime = down->second;
    unsigned long upDataCnt = up->first;
    unsigned long downDataCnt = down->first;
    double realTime = 
        downTime + (upTime - downTime) * (dataCnt - (double)downDataCnt) / ((double)upDataCnt - (double)downDataCnt);
    return realTime;
 
 }




void PipeProfiler::TranProfiler::train(){
    // param space
    int end = p->pgr_->pageNum_;
    end = std::min(end, p->pc_->bnlist);
    //printf("end:%d\n", end);
    int i = 1;

    std::vector<int> pages;
    std::vector<double> perf; 

    bool doubleone = false;

    while (i <= end){
        if(doubleone || i!=1)
            pages.push_back(i);

        auto t0 = timepoint();


        // allocate memory
        faiss::gpu::MemBlock mb = p->pgr_->allocMemory(i);

        FAISS_ASSERT(mb.valid == true);

        // radomly set allocated pages to clusters
        // and free these pages
        for (int j = 0; j < mb.pages.size(); j++){
            int clus = j;
            p->pgr_->pageinfo[mb.pages[j]] = clus;
            p->pc_->setonDevice(clus, mb.pages[j], true);

            // Memory transfer
            p->pgr_->memcpyh2d(mb.pages[j]);

            p->pc_->addGlobalCount(clus, mb.pages[j], 1);
        }

        double totalTime = timepoint() - t0;

        tranTimeDict[i] = totalTime;
        
        // Free these pages
        for (int j = 0; j < mb.pages.size(); j++){
            int clus = j;
            p->pgr_->pageinfo[mb.pages[j]] = -1;
            p->pc_->setonDevice(clus, mb.pages[j], false);
            p->pgr_->freetree_->insert(mb.pages[j], mb.pages[j]);
        }
        
        if(doubleone || i!=1){
            i = i << 1;
        }
        else{
            doubleone = true;
        }
    }


    istrained = true;
}

int PipeProfiler::decideSplit(int queryCnt, int dataCnt){
        int split = 1;
        // Find an appropriate split num
        if (pc_->Min_Block > 0){
            while(dataCnt * split < pc_->Min_Block){
                split = split << 1;
            }
        }
        return split;
    }


void PipeProfiler::ComProfiler::train(){
    // param space

    int ntmax = p -> nqMax;
    int d = p->pc_->d;
    int k = 10;


    std::mt19937 rng;
    float *trainvecs = new float[ntmax * d];
    {
        std::uniform_real_distribution<> distrib;
        for (size_t i = 0; i < ntmax * d; i++) {
            trainvecs[i] = distrib(rng);
        }
    }
    auto pc = p->pc_;
    auto pgr = p->pgr_;
    auto index = p->index_;
    auto device = index->ivfPipeConfig_.device;
    auto h2d_stream = pgr->getCopyH2DStream(device);
    auto exe_stream = pgr->getExecuteStream(device);
    DeviceScope scope(device);

    std::vector<void*> ListDataP_vec;
    std::vector<void*> ListIndexP_vec;//fake index
    std::vector<int> ListLength_vec;

    int listNum = std::min(pc->bnlist, (int)pgr->pageNum_);

    faiss::gpu::MemBlock mb = p->pgr_->allocMemory(listNum);

    FAISS_ASSERT(mb.valid == true);

    ListDataP_vec.resize(listNum);
    ListIndexP_vec.resize(listNum);
    ListLength_vec.resize(listNum);

    // radomly set allocated pages to clusters
    // and free these pages
    for (int j = 0; j < mb.pages.size(); j++){
        int clus = j;
        p->pgr_->pageinfo[mb.pages[j]] = clus;
        p->pc_->setonDevice(clus, mb.pages[j], true);

        // Memory transfer
        p->pgr_->memcpyh2d(mb.pages[j]);
        ListDataP_vec[j] = pgr->getPageAddress(mb.pages[j]);
        ListIndexP_vec[j] = (void *)((float*)(ListDataP_vec[j]) + 
                pc->d * pc->BCluSize[clus]);
        ListLength_vec[j] = pc->BCluSize[clus];
    }


    faiss::gpu::PipeTensor<void*, 1, true> ListDataP_({listNum}, pc);
    ListDataP_.copyFrom(ListDataP_vec, h2d_stream);
    ListDataP_.setResources(pc, pgr);
    ListDataP_.memh2d(h2d_stream);

    void** ListDataP = ListDataP_.devicedata();

    faiss::gpu::PipeTensor<int, 1, true> ListLength_({listNum}, pc);
    ListLength_.copyFrom(ListLength_vec, h2d_stream);
    ListLength_.setResources(pc, pgr);
    ListLength_.memh2d(h2d_stream);

    int* ListLength = ListLength_.devicedata();

    faiss::gpu::PipeTensor<void*, 1, true> ListIndexP_({listNum}, pc);
    ListIndexP_.copyFrom(ListIndexP_vec, h2d_stream);
    ListIndexP_.setResources(pc, pgr);
    ListIndexP_.memh2d(h2d_stream);

    void** ListIndexP = ListIndexP_.devicedata();

    int* queryids = (int*)malloc(sizeof(int) * ntmax);
    for (int i = 0; i < ntmax; i++){
        queryids[i] = i;
    }

    int nq = 16;

    faiss::gpu::PipeTensor<int, 1, true> queryids_gpu({nq}, pc);
    queryids_gpu.copyFrom(queryids, h2d_stream);
    queryids_gpu.setResources(pc, pgr);
    queryids_gpu.memh2d(h2d_stream);

        
    faiss::gpu::PipeTensor<float, 2, true> queries_gpu({nq, d}, pc);
    queries_gpu.copyFrom(trainvecs, h2d_stream);
    queries_gpu.setResources(pc, pgr);
    queries_gpu.memh2d(h2d_stream);

    bool dir;
    if (p->index_->metric_type == faiss::MetricType::METRIC_L2) {
        faiss::gpu::L2Distance metr;
        dir = metr.kDirection;                                            
    } else if (p->index_->metric_type == faiss::MetricType::METRIC_INNER_PRODUCT) {
        faiss::gpu::IPDistance metr;          
        dir = metr.kDirection;
    }

    // query*cluster
    
    int maxClus = p->maxDataCnt / nq;

    int clus = 1;
    while (clus <= maxClus)
    {

        int* query_bcluster_matrix = new int[2 * clus * nq];
        std::default_random_engine generator;
        std::normal_distribution<double> distribution(clus, 0.8 * (double)clus);

        for (int i = 1; i < nq; i += 2) {
            int number = rand() % (2 * clus);//(int)distribution(generator);
            if(number >= 2 * clus){
                number = 2 * clus - 1;
            }
            else if(number == 0){
                number = 1;
            }
            for(int j = 0 ; j < 2 * clus ; j++){
                if(j < number){
                    query_bcluster_matrix[i * 2 * clus + j] = j % std::min(clus, listNum);
                }
                else{
                    query_bcluster_matrix[i * 2 * clus + j] = -1;
                }
            }
            for(int j = 0 ; j < 2 * clus ; j++){
                if(j < 2 * clus - number){
                    query_bcluster_matrix[(i - 1)  * 2 * clus + j] = j % std::min(clus, listNum);
                }
                else{
                    query_bcluster_matrix[(i - 1) * 2 * clus + j] = -1;
                }
            }
                
        }
        if(nq % 2 == 1){
            for(int j = 0 ; j < 2 * clus ; j++){
                if(j < clus){
                    query_bcluster_matrix[(nq - 1)  * 2 * clus + j] = j % std::min(clus, listNum);
                }
                else{
                    query_bcluster_matrix[(nq - 1) * 2 * clus + j] = -1;
                }
            }
        }
        faiss::gpu::PipeTensor<int, 2, true> query_cluster_matrix_gpu({nq, 2 * clus}, pc);
        query_cluster_matrix_gpu.copyFrom(query_bcluster_matrix, h2d_stream);
        query_cluster_matrix_gpu.setResources(pc, pgr);
        query_cluster_matrix_gpu.memh2d(h2d_stream);
        hipStreamSynchronize(h2d_stream);

        int dataCnt = nq * clus;
        int split = p->decideSplit(nq, dataCnt);

        faiss::gpu::PipeTensor<float, 2, true> out_distances({nq, (int)k}, pc);
        out_distances.setResources(pc, pgr);
        out_distances.reserve();

        faiss::gpu::PipeTensor<int, 2, true> out_indices({nq, (int)k}, pc);
        out_indices.setResources(pc, pgr);
        out_indices.reserve();

        double t0 = timepoint();

        faiss::gpu::runKernelComputeReduce(
                        d,
                        k,
                        nq,
                        2 * clus,
                        queryids_gpu,
                        queries_gpu,
                        query_cluster_matrix_gpu,
                        ListDataP,
                        p->index_->ivfPipeConfig_.indicesOptions,
                        ListLength,
                        ListIndexP,
                        p->index_->metric_type,
                        dir,
                        out_distances,
                        out_indices,
                        pc,
                        pgr,
                        device,
                        split);

        hipStreamSynchronize(exe_stream);

        double t1 = timepoint();
        double tCnt = t1 - t0;
        computeTimeDict[dataCnt] = tCnt;
        clus *= 2 ;
        printf("query:%d, dataCnt:%d, split:%d. Result:%lf\n", nq, dataCnt, split, tCnt);
        delete[] query_bcluster_matrix;
        }

    
    for (int j = 0; j < mb.pages.size(); j++){
        int clus = j;
        p->pgr_->pageinfo[mb.pages[j]] = -1;
        p->pc_->setonDevice(clus, mb.pages[j], false);
        p->pgr_->freetree_->insert(mb.pages[j], mb.pages[j]);
    }
    
    delete[] trainvecs;
    
    free(queryids);    
    istrained = true;
}



} // namespace gpu
} // namespace faiss