/**
 * Copyright (c) zili zhang & fangyue liu @PKU.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

/**
 * Copyright (c) Facebook, Inc. and its affiliates.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <iostream>
#include <vector>
#include <list>
#include <algorithm>

#include <sys/stat.h>
#include <sys/types.h>
#include <sys/time.h>
#include <unistd.h>
#include <omp.h>
#include <assert.h>

#include <faiss/gpu/GpuAutoTune.h>
#include <faiss/gpu/GpuCloner.h>
#include <faiss/gpu/GpuIndexFlat.h>
#include <faiss/gpu/GpuIndexIVFFlat.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/index_io.h>

double elapsed() {
    struct timeval tv;
    gettimeofday(&tv, nullptr);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}


bool file_exist(const std::string& file_path)
{
	if (FILE* file = fopen(file_path.c_str(), "r")){
		fclose(file);
		return true;
	}
	else 
		return false;
}

double inter_sec(faiss::Index::idx_t *taget, int *gt, int k){
    double res = 0.;
    for (int i = 0; i < k; i++){
        int val = taget[i];
        for (int j = 0; j < k; j++){
            if (val == gt[j]){
                res += 1.;
                break;
            }
        }
    }
    return res / k;
}

float* fvecs_read(const char* fname, size_t* d_out, size_t* n_out) {
    FILE* f = fopen(fname, "r");
    if (!f) {
        fprintf(stderr, "could not open %s\n", fname);
        perror("");
        abort();
    }
    int d;
    fread(&d, 1, sizeof(int), f);
    assert((d > 0 && d < 1000000) || !"unreasonable dimension");
    fseek(f, 0, SEEK_SET);
    struct stat st;
    fstat(fileno(f), &st);
    size_t sz = st.st_size;
    assert(sz % ((d + 1) * 4) == 0 || !"weird file size");
    size_t n = sz / ((d + 1) * 4);

    *d_out = d;
    *n_out = n;
    float* x = new float[n * (d + 1)];
    size_t nr = fread(x, sizeof(float), n * (d + 1), f);
    assert(nr == n * (d + 1) || !"could not read whole file");

    // shift array to remove row headers
    for (size_t i = 0; i < n; i++)
        memmove(x + i * d, x + 1 + i * (d + 1), d * sizeof(*x));

    fclose(f);
    return x;
}

std::vector<float *> fvecs_reads(const char* fname, size_t* d_out, size_t* n_out, int slice = 10){
    FILE* f = fopen(fname, "r");
    if (!f) {
        fprintf(stderr, "could not open %s\n", fname);
        perror("");
        abort();
    }
    int d;
    fread(&d, 1, sizeof(int), f);
    assert((d > 0 && d < 1000000) || !"unreasonable dimension");
    fseek(f, 0, SEEK_SET);
    struct stat st;
    fstat(fileno(f), &st);
    size_t sz = st.st_size;
    assert(sz % ((d + 1) * 4) == 0 || !"weird file size");
    size_t n = sz / ((d + 1) * 4);

    *d_out = d;
    *n_out = n;
    std::vector<float *> res;
    size_t nr = 0;
    size_t slice_size = n / slice * (d + 1);
    size_t total_size = size_t(d + 1) * size_t(n);

    for (int i = 0; i < slice; i++){
        float* x = new float[slice_size];
        nr += fread(x, sizeof(float), slice_size, f);
        for (size_t j = 0; j < n / slice; j++)
            memmove(x + j * d, x + 1 + j * (d + 1), d * sizeof(*x));
        res.push_back(x);
    }

    assert(nr == total_size || !"could not read whole file");
    fclose(f);
    return res;
}

// not very clean, but works as long as sizeof(int) == sizeof(float)
int* ivecs_read(const char* fname, size_t* d_out, size_t* n_out) {
    return (int*)fvecs_read(fname, d_out, n_out);
}

std::vector<float*> fbin_reads(const char* fname, size_t* d_out, size_t* n_out, int slice = 100) {
    std::vector<float*> vec(slice);
    FILE* f = fopen(fname, "r");
    int d, n;
    fread(&n, sizeof(int), 1, f);
    fread(&d, sizeof(int), 1, f);
    fclose(f);
    printf("d : %d, n: %d\n", d, n);
    assert((d > 0 && d < 1000000) || !"unreasonable dimension");
    *d_out = d;
    *n_out = n;
    int64_t total_size = int64_t(d) * int64_t(n);
    int64_t slice_size = total_size / slice;
    int num = 0;
#pragma omp parallel for
    for (int i = 0; i < slice; i++){
        auto t0 = elapsed();
        FILE* f = fopen(fname, "r");
        if (!f) {
            fprintf(stderr, "could not open %s\n", fname);
            perror("");
            abort();
        }
        int64_t nr = 0;
        int64_t start = slice_size * i * sizeof(float) + 8;
        fseek(f, start, SEEK_SET);
        float *x = new float[slice_size];
        nr += fread(x, sizeof(float), slice_size, f);
        vec[i] = x;
        auto t1 = elapsed();
        int id = omp_get_thread_num();
        #pragma critical
        {
            printf("Read %d/%d slice done... , Thread %d : %.3f s\n", i, slice, id, t1 - t0);
            printf("Read %d/%d done\n", num++, slice);
        }

        // int64_t nr = fread(x, sizeof(float), total_size, f);
        // printf("Read finished, read %ld\n", nr);
        // assert(nr == total_size || !"could not read whole file");
        fclose(f);
    }
    return vec;
}

// ./script dataset-name bs topk (./overall deep 256 10)
int main(int argc,char **argv){
    std::cout << argc << " arguments" <<std::endl;
    if(argc - 1 != 3){
        printf("You should at least input 3 params: the dataset name, batch size and topk\n");
        return 0;
    }
    std::string p1 = argv[1];
    std::string p2 = argv[2];
    std::string p3 = argv[3];
    int input_k = std::stoi(p3);
    int bs = std::stoi(p2);
    int ncentroids = -1;

    std::string db, train_db, query, gtI, gtD;
    int dim;
    if (input_k>100 || input_k <=0){
        printf("Input topk must be lower than or equal to 100 and greater than 0\n");
        return 0;
    }
    if (p1 == "sift"){
        db = "/billion-data/data2/sift1B.fbin";
        train_db = "/billion-data/data4/sift/sift10M/sift10M.fvecs";
        query = "/billion-data/data4/sift/sift10M/query.fvecs";
        gtI = "/billion-data/data2/sift1Bgti.ivecs";
        gtD = "/billion-data/data2/sift1Bgtd.fvecs";
        dim = 128;
        ncentroids = 1921;
    }
    else if (p1 == "deep"){
        db = "/billion-data/data1/deep1B.fbin";
        train_db = "/billion-data/data4/deep/deep10M.fvecs";
        query = "/billion-data/data4/deep/query.fvecs";
        gtI = "/billion-data/data1/deep1Bgti.ivecs";
        gtD = "/billion-data/data1/deep1Bgtd.fvecs";
        dim = 96;
        ncentroids = 2845;
    }
    else if (p1 == "text"){
        db = "/billion-data/data3/text1B.fbin";
        train_db = "/billion-data/data4/text/text10M.fvecs";
        query = "/billion-data/data4/text/query.fvecs";
        gtI = "/billion-data/data3/text1Bgti.ivecs";
        gtD = "/billion-data/data3/text1Bgtd.fvecs";
        dim = 200;
        ncentroids = 1313;
    }
    else{
        printf("Your input dataset is not included yet! \n");
        return 0;
    }

    auto t0 = elapsed();

    omp_set_num_threads(64);

    int dev_no = 0;
    faiss::gpu::StandardGpuResources resources;
    faiss::gpu::GpuIndexIVFFlatConfig config;
    config.device = dev_no;
    faiss::gpu::GpuIndexIVFFlat *index;

    if (p1 == "text" || p1 == "text30"){
        index = new faiss::gpu::GpuIndexIVFFlat(
            &resources, dim, ncentroids, faiss::METRIC_INNER_PRODUCT, config);
    }
    else{
        index = new faiss::gpu::GpuIndexIVFFlat(
            &resources, dim, ncentroids, faiss::METRIC_L2, config);
    }

    size_t d;
    // Train the index
    {
        printf("[%.3f s] Loading train set\n", elapsed() - t0);

        size_t nt;
        float* xt = fvecs_read(train_db.c_str(), &d, &nt);

        FAISS_ASSERT(d == dim);
        printf("[%.3f s] Training on %ld vectors\n", elapsed() - t0, nt);

        index->train(nt, xt);
        delete[] xt;
    }

    // Add the data
    {
        printf("[%.3f s] Loading database\n", elapsed() - t0);

        size_t nb, d2;
        int slice = 100;
        omp_set_num_threads(8);
        // std::vector<float *> xbs = fvecs_reads(db.c_str(), &d2, &nb, slice);
        std::vector<float *> xbs = fbin_reads(db.c_str(), &d2, &nb, slice);
        omp_set_num_threads(64);
        assert(d == d2 || !"dataset does not have same dimension as train set");

        printf("[%.3f s] Indexing database, size %ld*%ld\n",
               elapsed() - t0,
               nb,
               d);

        for (int i = 0; i < slice; i++){
            double tt0 = elapsed();
            index->add(nb / slice, xbs[i]);
            delete[] xbs[i];
            double tt1 = elapsed();
            printf("Index %d/%d done : %.3f s\n", i, slice, tt1 - tt0);
        }
    }

    size_t nq;
    float* xq;
    {
        printf("[%.3f s] Loading queries\n", elapsed() - t0);

        size_t d2;
        xq = fvecs_read(query.c_str(), &d2, &nq);
        assert(d == d2 || !"query does not have same dimension as train set");
    }

    size_t k;                // nb of results per query in the GT
    int* gt; // nq * k matrix of ground-truth nearest-neighbors

    {
        printf("[%.3f s] Loading ground truth for %ld queries\n",
               elapsed() - t0,
               nq);

        // load ground-truth and convert int to long
        size_t nq2;
        int* gt_int = ivecs_read(gtI.c_str(), &k, &nq2);
        assert(nq2 == nq || !"incorrect nb of ground truth entries");

        gt = new int[k * nq];
        for (int i = 0; i < k * nq; i++) {
            gt[i] = gt_int[i];
        }
        delete[] gt_int;
    }

    float *gtd;
    {
        printf("[%.3f s] Loading ground truth for %ld queries\n",
               elapsed() - t0,
               nq);

        // load ground-truth and convert int to long
        size_t nq2;
        gtd = fvecs_read(gtD.c_str(), &k, &nq2);
        assert(nq2 == nq || !"incorrect nb of ground truth entries");
    }

    if(bs == 1){
        nq = 300;
    }
    else{
        nq = 2560;
    }
    // Start queries
    std::vector<float> dis(nq * input_k);
    std::vector<faiss::Index::idx_t> idx(nq * input_k);
    index->nprobe = ncentroids / 8;

    auto tt0 = elapsed();
    int i;
    for (i = 0; i < nq / bs; i++){
        auto tmpt0 = elapsed();
        index->search(bs, xq + d * (bs * i), input_k, dis.data() + input_k * (bs * i), idx.data() + input_k * (bs * i));
        auto tmpt1 = elapsed();
        printf("%d=th batch cost %.3f s\n", i, tmpt1 - tmpt0);
    }
    auto tt1 = elapsed();
    double total = tt1 - tt0;

    double acc = 0.;
    for (int j = 0; j < i * bs; j++){
        auto tmp = inter_sec(idx.data() + input_k * j, gt + k * j, input_k);
        acc += tmp;
    }
    acc /= (i * bs);
    acc *= 100;

    printf("Ave Latency : %.3f s\n", total / i);
    printf("Ave accuracy : %.1f%% \n", acc);

    // hipFree(x);

    delete[] xq;
    delete[] gt;
    delete[] gtd;
    delete index;

    return 0;
}