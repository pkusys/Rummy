#include "hip/hip_runtime.h"
/**
 * Copyright (c) zili zhang & fangyue liu @PKU.
 *
 * This source code is licensed under the MIT license found in the
 * LICENSE file in the root directory of this source tree.
 */

#include <cmath>
#include <cstdio>
#include <cstdlib>
#include <random>
#include <cassert>
#include <cstring>

#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>

#include <sys/time.h>
#include <omp.h>
#include <cinttypes>
#include <stdint.h>
#include <algorithm>
#include <mutex>
#include <string.h>
#include <limits>
#include <memory>

#include <omp.h>

#include <faiss/pipe/IndexIVFPipe.h>
#include <faiss/gpu/StandardGpuResources.h>
#include <faiss/gpu/PipeGpuResources.h>
#include <faiss/gpu/utils/PipeTensor.cuh>
#include <faiss/pipe/PipeScheduler.h>
#include <faiss/impl/FaissAssert.h>
#include <faiss/pipe/PipeKernel.cuh>

double elapsed() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

bool file_exist(const std::string& file_path)
{
	if (FILE* file = fopen(file_path.c_str(), "r")){
		fclose(file);
		return true;
	}
	else 
		return false;
}

double inter_sec(int *taget, int *gt, int k){
    double res = 0.;
    for (int i = 0; i < k; i++){
        int val = taget[i];
        for (int j = 0; j < k; j++){
            if (val == gt[j]){
                res += 1.;
                break;
            }
        }
    }
    return res / k;
}

float* fvecs_read(const char* fname, size_t* d_out, size_t* n_out) {
    FILE* f = fopen(fname, "r");
    if (!f) {
        fprintf(stderr, "could not open %s\n", fname);
        perror("");
        abort();
    }
    int d;
    fread(&d, 1, sizeof(int), f);
    assert((d > 0 && d < 1000000) || !"unreasonable dimension");
    fseek(f, 0, SEEK_SET);
    struct stat st;
    fstat(fileno(f), &st);
    size_t sz = st.st_size;
    assert(sz % ((d + 1) * 4) == 0 || !"weird file size");
    size_t n = sz / ((d + 1) * 4);

    *d_out = d;
    *n_out = n;
    float* x = new float[n * (d + 1)];
    size_t nr = fread(x, sizeof(float), n * (d + 1), f);
    assert(nr == n * (d + 1) || !"could not read whole file");

    // shift array to remove row headers
    for (size_t i = 0; i < n; i++)
        memmove(x + i * d, x + 1 + i * (d + 1), d * sizeof(*x));

    fclose(f);
    return x;
}

// not very clean, but works as long as sizeof(int) == sizeof(float)
int* ivecs_read(const char* fname, size_t* d_out, size_t* n_out) {
    return (int*)fvecs_read(fname, d_out, n_out);
}

int main(){
    omp_set_num_threads(8);
    auto t0 = elapsed();

    int dim = 96;
    int dev_no = 0;
    int ncentroids = 64 * 4;
    
    faiss::gpu::PipeGpuResources* pipe_res = new faiss::gpu::PipeGpuResources();
    faiss::IndexIVFPipeConfig config;
    faiss::IndexIVFPipe* index = new faiss::IndexIVFPipe(dim, ncentroids, config, pipe_res, faiss::METRIC_L2);
    // faiss::IndexIVFPipe* index = new faiss::IndexIVFPipe(dim, ncentroids, config, pipe_res, faiss::METRIC_INNER_PRODUCT);

    FAISS_ASSERT (config.interleavedLayout == true);

    size_t d;
    // Train the index
    {
        printf("[%.3f s] Loading train set\n", elapsed() - t0);

        size_t nt;
        float* xt = fvecs_read("/workspace/data/deep/deep10M.fvecs", &d, &nt);

        FAISS_ASSERT(d == dim);

        printf("[%.3f s] Training on %ld vectors\n", elapsed() - t0, nt);

        nt = 10000000;

        index->train(nt, xt);
        delete[] xt;
    }

    // Add the data
    {
        printf("[%.3f s] Loading database\n", elapsed() - t0);

        size_t nb, d2;
        float* xb = fvecs_read("/workspace/data/deep/deep10M.fvecs", &d2, &nb);
        assert(d == d2 || !"dataset does not have same dimension as train set");

        printf("[%.3f s] Indexing database, size %ld*%ld\n",
               elapsed() - t0,
               nb,
               d);

        index->add(nb, xb);

        delete[] xb;
    }

    size_t nq;
    float* xq;
    {
        printf("[%.3f s] Loading queries\n", elapsed() - t0);

        size_t d2;
        xq = fvecs_read("/workspace/data/deep/query.fvecs", &d2, &nq);
        assert(d == d2 || !"query does not have same dimension as train set");
    }

    size_t k;                // nb of results per query in the GT
    int* gt; // nq * k matrix of ground-truth nearest-neighbors

    {
        printf("[%.3f s] Loading ground truth for %ld queries\n",
               elapsed() - t0,
               nq);

        // load ground-truth and convert int to long
        size_t nq2;
        int* gt_int = ivecs_read("/workspace/data/deep/idx.ivecs", &k, &nq2);
        assert(nq2 == nq || !"incorrect nb of ground truth entries");

        gt = new int[k * nq];
        for (int i = 0; i < k * nq; i++) {
            gt[i] = gt_int[i];
        }
        delete[] gt_int;
    }

    float *gtd;
    {
        printf("[%.3f s] Loading ground truth for %ld queries\n",
               elapsed() - t0,
               nq);

        // load ground-truth and convert int to long
        size_t nq2;
        gtd = fvecs_read("/workspace/data/deep/dis.fvecs", &k, &nq2);
        assert(nq2 == nq || !"incorrect nb of ground truth entries");
    }
    printf("[%.3f s] Start Balancing\n",
               elapsed() - t0);
    index->balance();
    printf("[%.3f s] Finishing Balancing: %d B clusters\n",
               elapsed() - t0, index->pipe_cluster->bnlist);

    auto pc = index->pipe_cluster;
    pipe_res->initializeForDevice(0, pc);

    printf("[%.3f s] Start Profile\n",
               elapsed() - t0);
    // Train profile
    std::string profile_name = "Profile_" + std::string("Sift_") + std::to_string(ncentroids) + ".txt";
    if (!file_exist(profile_name.c_str())){
        index->profile();
        index->saveProfile(profile_name.c_str());
    }
    else{
        index->loadProfile(profile_name.c_str());
    }
    printf("[%.3f s] Finish Profile\n",
               elapsed() - t0);

    int bs = 256;
    int topk = 10;
    std::vector<float> dis(bs * topk);
    std::vector<int> idx(bs * topk);
    index->set_nprobe(ncentroids / 8);

    auto tt0 = elapsed();

    // std::cout << pc->PinTempStatus() << "\n";
    // std::cout << pipe_res->tempMemory_[0]->toString() << "\n";

    auto sche = new faiss::gpu::PipeScheduler(index, 
            pc, pipe_res, bs, xq, topk, dis.data(), idx.data());
    auto tt1 = elapsed();
    printf("Search Time: %.3f ms\n", (tt1 - tt0)*1000);
    printf("Computation Time: %.3f ms, Transmission Time: %.3f ms\n", 
        sche->com_time*1000, sche->com_transmission*1000);
    delete sche;

    for (int i = 0; i < topk; i++){
        printf("%d %ld: %f %f\n", idx[i + topk * 128], gt[i + 100 * 128], dis[i + topk * 128], gtd[i + 100 * 128]);
    }

    // std::cout << pc->PinTempStatus() << "\n";
    // std::cout << pipe_res->tempMemory_[0]->toString() << "\n";

    printf("\n--- Next Batches ---\n");
    index->set_nprobe(ncentroids / 16);
    double total = 0.;
    double acc = 0.;
    int newbs = 1;
    int size = 50;
    double ave_opt = 0.;
    for (int i = 0; i < size; i++){
        tt0 = elapsed();
        sche = new faiss::gpu::PipeScheduler(index, 
                pc, pipe_res, newbs, xq + d * (bs + newbs*i), topk, dis.data(), idx.data());
        tt1 = elapsed();
        printf("Second Search Time: %.3f ms\n", (tt1 - tt0)*1000);
        total += tt1 - tt0;
        printf("Computation Time: %.3f ms, Transmission Time: %.3f ms\n", 
            sche->com_time*1000, sche->com_transmission*1000);
        ave_opt += std::max(sche->com_time*1000, sche->com_transmission*1000);
        delete sche;
        for (int j = 0; j < newbs; j++)
            acc += inter_sec(idx.data() + topk * j, gt + k * ((bs + newbs*i) + j), topk);
    }

    printf("Ave Opt Latency : %.3f ms\n", ave_opt / size);
    printf("Ave Latency : %.3f ms\n", total * 1000. / size);
    printf("Ave accuracy : %.1f%% \n", acc * 100. / (size * newbs));

    // std::cout << pc->PinTempStatus() << "\n";
    // std::cout << pipe_res->tempMemory_[0]->toString() << "\n";


    delete[] xq;
    delete[] gt;
    delete[] gtd;
    delete index;
    delete pipe_res;
    return 0;
}